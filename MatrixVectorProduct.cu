#include "hip/hip_runtime.h"
#include "VectorMath.hh"

#include "helper.hh"
#include <vector>

#include <iostream>

using namespace std; 

#define THREADS_PER_BLOCK 1024

__global__ void MVP_kernel(double * A, double * x, double * b, int N) {

	int tid = blockIdx.x * blockDim.x + threadIdx.x; 

	double sum = 0; 

	for (int i=0; i<N; i++) {

		int ind = tid*N + i; 

		sum += A[ind] * x[i]; 

	}

	b[tid] = sum; 

}

vector<double> MVP_GPU(vector<vector<double>> &A, vector<double> &x) {

	int N = A.size(); 

	size_t dSize = sizeof(double); 

	// device pointers 
	double *d_A, *d_x, *d_b;
	double *b; 

	b = new double[N]; 

	hipMalloc(&d_A, N*N*dSize); 
	hipMalloc(&d_x, N*dSize); 
	hipMalloc(&d_b, N*dSize); 

	double * array = new double[N*N]; 

	// convert from vector to array 
	for (int i=0; i<N; i++) {

		for (int j=0; j<N; j++) {

			int ind = N*i + j; 

			array[ind] = A[i][j]; 

		}

	}

	hipMemcpy(d_A, array, N*N*dSize, hipMemcpyHostToDevice); 
	hipMemcpy(d_x, &x[0], N*dSize, hipMemcpyHostToDevice); 

	MVP_kernel<<<N/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_A, d_x, d_b, N); 

	hipMemcpy(b, d_b, N*dSize, hipMemcpyDeviceToHost);

	// convert to vector 
	vector<double> b_vec(N); 
	for (int i=0; i<N; i++) {

		b_vec[i] = b[i]; 

	}

	hipFree(d_A); hipFree(d_x); hipFree(d_b); 

	delete b; delete array; 

	return b_vec; 

}